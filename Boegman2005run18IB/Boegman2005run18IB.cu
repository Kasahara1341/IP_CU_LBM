#include "hip/hip_runtime.h"

#include "../all.hpp"

__device__ float profile_s22(float xi, float r){
    float result =    (0 * ( r > xi/2.0 )) 
                    + (0.5*(sin(3.14159*r/xi) + 1.0) * ( fabsf(r) <= xi/2.0 )) 
                    + (1.0 * ( r < -xi/2.0 )) ;
    return result ;
}
__global__ void SPM_ellipse3D(float *items, float *f, float *posx, float *posy, float *posz, float *velx, float *vely, float *velz, float *velB){
    // smoothed-profile method
    int id_rho = blockIdx.x * blockDim.x + threadIdx.x ;
    int id_f = id_rho * (int)items[IDX_Q] ;
    if(id_rho<items[IDX_num_calc]){
        float distance = (0.15*posx[id_rho]+posz[id_rho]-0.29)/sqrtf(1+powf(0.15,2)) ;
        float fx=0, fy=0, fz=0 ;
        fx = (velB[0]*0 - velx[id_rho])/items[IDX_dt] * profile_s22(items[IDX_dz],distance) ;
        fy = (velB[1]*0 - vely[id_rho])/items[IDX_dt] * profile_s22(items[IDX_dz],distance) ;
        fz = (velB[2]*0 - velz[id_rho])/items[IDX_dt] * profile_s22(items[IDX_dz],distance) ;
        for(int k =0;k<items[IDX_Q];k++){
            float tmp = items[IDX_w(k)]*items[IDX_dt] * 3.0
            *( items[IDX_cx(k)]*fx + items[IDX_cy(k)]*fy + items[IDX_cz(k)]*fz )/(powf(items[IDX_c],2)) ;
            f[id_f+k] += tmp ;
            velx[id_rho] += items[IDX_cx(k)] * tmp ;
            vely[id_rho] += items[IDX_cy(k)] * tmp ;
            velz[id_rho] += items[IDX_cz(k)] * tmp ;
        }
    }
}
template<typename Typ>
__global__ void update_velIBM2(Typ *items, Typ *f, Typ *velx, Typ *vely, Typ *velz, Typ *FIBx, Typ *FIBy, Typ *FIBz){
    int id_rho = blockIdx.x * blockDim.x + threadIdx.x ;
    if(id_rho<items[IDX_num_calc]){
        int id_f = id_rho * (int)items[IDX_Q] ;
        for(int k =0;k<items[IDX_Q];k++){
            Typ tmp = items[IDX_dt]*items[IDX_w(k)]*(items[IDX_cx(k)]*FIBx[id_rho] + items[IDX_cy(k)]*FIBy[id_rho] + items[IDX_cz(k)]*FIBz[id_rho]) ;
            f[id_f+k]    += tmp ;
            velx[id_rho] += items[IDX_cx(k)] * tmp ;
            vely[id_rho] += items[IDX_cy(k)] * tmp ;
            velz[id_rho] += items[IDX_cz(k)] * tmp ;
        } 
    }
}
float sech(float x){
    return 2.0/(exp(x)+exp(-x)) ;
}


int main (void){

    // h1 is upper layer. h2 is lower layer
    float H=0.29, h1, h2, eta0, c0, alpha, thicness=0.01 ; 
    h1=H*0.50 ; h2=H-h1 ; eta0 =  h1*0.82 ;
    c0 = sqrt(9.81*20/1020*(h1*h2)/H) ;
    alpha = 1.5 * c0 * (h1-h2)/(h1*h2) ;
    float Ti = 2*6.0/c0, Ts = 6.0/(alpha*eta0) ;
    float slope_eta0 = eta0 / 3.0 , slope= 3.0/20.0 ;

    int i, j, l, k ;
    int *cx, *cy, *cz ;
    vector<int> neib, nextK, nextB, divx, divy ; // devided x, y
    vector<float> f, g, h, Fk, vel_x, vel_y, vel_z, rho, sal, phi, pressure, Fx, Fy, Fz, posx, posy, posz, delX, delY ;
    vector<float> tau, taus ;
    vector<float> item ;
    Items items ; input_items(items,"./input/initial") ; 
    items.PFthick = 3.5*items.dx ; items.sigma = 0.072 *0;
    string Boussinesq_approxi = "on" ; 
    bool Boussi_flag = (strcmp(Boussinesq_approxi.c_str(),"on") ==0) ;
    items.setc(9) ; 
    vector<float> M((int)pow(items.num_velocity,2)), MM((int)pow(items.num_velocity,2)),
    M_inv((int)pow(items.num_velocity,2)), S(items.num_velocity) ;
    set_M<float>(items.num_velocity, M, S, M_inv, MM) ;

    ////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////
    items.save_interval = 1.0/items.dt ; items.total_count= 75/items.dt ;
    items.save_interval = items.total_count/150 ;
    // items.total_count=20 ; items.save_interval=1 ;
    ////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////
        
    cx = new int[items.num_velocity] ; cy= new int[items.num_velocity] ; cz = new int[items.num_velocity] ; 
    for(k=0;k<items.num_velocity;k++){
        cx[k]=items.cx[k]/items.c ; cy[k]=items.cy[k]/items.c ; cz[k]=items.cz[k]/items.c ;
    }

    // divide x, y direction
    for(i=0;i<items.nx;i++){
        float x = (i+0.5)*items.dx*items.ratiox ;
        if( x < 3.0 ){
            if( x < 2.5){
                if(x< 2.0){
                    divx.push_back(30) ; continue ;
                }
                divx.push_back(10) ; continue ;
            }
            divx.push_back(6) ; continue ;
            // divx.push_back(1) ;
        }
        else{divx.push_back(1);}
    } // */
    // divide x, y direction
    // for(i=0;i<items.nx;i++){
        // divx.push_back(1) ;
    // }

    for(j=0;j<items.ny;j++){divy.push_back(1) ; }
    // set wall infomation
    vector<int> lnum ;
    items.num_calc=0 ;
    for(i=0;i<items.nx;i++){
        for(int divi=0;divi<divx[i];divi++){
            for(j=0;j<items.ny;j++){
                for(int divj=0;divj<divy[j];divj++){
                    for(l=0;l<items.nz;l++){
                        float x = ( i + (2.0*divi+1.0)/(2.0*divx[i]) )*items.dx*items.ratiox ;
                        float y = ( j + (2.0*divj+1.0)/(2.0*divy[j]) )*items.dx*items.ratioy ;
                        float z = ( l + 0.5 )*items.dx ;
                        // if(z > 0.29 -6*items.dx - slope * x){
                        if(x > 0){
                            lnum.push_back(items.num_calc) ;
                            items.num_calc+=1 ;
                        }
                        else{lnum.push_back(-1) ; }
                    }
                }
            }
        }
    }    

    printf("set initial condition \n") ;
    float rhoL=1, rhoH=1000.0, muL=1.*pow(10,-5), muH=1.016*pow(10,-3) ;
    { int int_tmp=0 ;
    for(i=0;i<items.nx;i++){
        for(int divi=0;divi<divx[i];divi++){
            for(j=0;j<items.ny;j++){
                for(int divj=0;divj<divy[j];divj++){
                    for(l=0;l<items.nz;l++){
                        if(lnum[int_tmp]<0){ int_tmp+=1 ; continue ;}
                        float local_x = (i + (2*divi+1.0)/(2.0*divx[i]) )*items.dx*items.ratiox ;
                        float local_z = (0.5 + l)*items.dx ;
                        posx.push_back(local_x) ; posz.push_back(local_z) ;
                        posy.push_back( (j + (2*divj+1.0)/(2.0*divy[j]) )*items.dx*items.ratioy ) ;
                        delX.push_back( items.dx*items.ratiox/divx[i]) ; delY.push_back( items.dx*items.ratioy/divy[j]) ;
                        sal.push_back(0) ; phi.push_back(1) ;           
                        sal[sal.size()-1] = 12.13-12.13*tanh((local_z-(h2+(local_x-3)*slope_eta0))/thicness) ; // 6m 
                        // sal[sal.size()-1] = 12.13-12.13*tanh((local_z-(h2+(local_x-3)*slope_eta0*0))/thicness) ; // horizontal ditribution
                        if(posz[posz.size()-1] < 0.29 - slope * posx[posx.size()-1]){
                            sal[sal.size()-1] = 0 ;
                        }
                        // phi[phi.size()-1] = 0.5 - 0.5*tanh( (posz[posz.size()-1] - (d0+eta) )/items.PFthick*2.0 ) ; // 
                        rho.push_back(rhoL+phi[phi.size()-1]*(rhoH-rhoL) + sal[sal.size()-1]*0.824493) ;

                        // pressure.push_back(9.81*(items.dx*items.nz-z) *3.0/(pow(items.c,2))) ; // 静水圧を仮定したp*の分布 
                        pressure.push_back(0) ;
                        vel_x.push_back(0) ; vel_y.push_back(0) ; vel_z.push_back(0) ; 
                        Fx.push_back(0) ; Fy.push_back(0) ; Fz.push_back(0) ;
                        tau.push_back( 1.0*3.0*(muL+phi[phi.size()-1]*(muH-muL))/rho[rho.size()-1]/pow(items.c,2)/items.dt+0.5) ;
                        taus.push_back(1.4*3.0*pow(10,-9)/pow(items.c,2)/items.dt+0.5) ;
                        for(k=0;k<items.num_velocity;k++){
                            float tmp = (vel_x[vel_x.size()-1]*items.cx[k] + vel_y[vel_x.size()-1]*items.cy[k] + vel_z[vel_x.size()-1]*items.cz[k])/pow(items.c,2) ;
                            Fk.push_back(0) ; f.push_back(0) ;
                            g.push_back(items.weight[k]*sal[sal.size()-1]*(1 + 3*tmp + 4.5*tmp*tmp - 1.5*(pow(vel_x[vel_x.size()-1],2)+pow(vel_y[vel_x.size()-1],2)+pow(vel_z[vel_x.size()-1],2))/pow(items.c,2) )) ;
                        }
                        int_tmp+= 1 ;
                    }
                }
            }
        }
    } }
    items.nx=0 ; items.ny=0 ; 
    for(i=0;i<divx.size();i++){items.nx+=divx[i];} for(j=0;j<divy.size();j++){items.ny+=divy[j];}
    for(i=0;i<items.nx;i++){for(j=0;j<items.ny;j++){for(l=0;l<items.nz;l++){
        if(lnum[i*items.ny*items.nz+j*items.nz+l]<0){
            continue ;
        }
        for(k=0;k<items.num_velocity;k++){
            if(i+cx[k]>=0 && i+cx[k]<items.nx && j+cy[k]>=0 && j+cy[k]<items.ny && l+cz[k]>=0 && l+cz[k]<items.nz){
                neib.push_back(lnum[i*items.ny*items.nz+j*items.nz+l + cx[k]*items.ny*items.nz + cy[k]*items.nz + cz[k]]) ;
            }
            else{ neib.push_back(-1) ; }
        }
    }}}

    /* set neighbor wall lattice */
    printf("set neighbor wall lattice \n") ;
    set_neibghor_wall(items,lnum,divx,divy,neib,f,g,Fk,pressure,rho,phi,posx,posy,posz,delX,delY,vel_x,vel_y,vel_z) ;
    cout<<"items nx ="<<items.nx<<" items ny="<<items.ny<<endl<<endl;
    cout<<"number of calculation lattice is "<<items.num_calc<<" wall lattice is "<<rho.size()-items.num_calc<<endl; cout<<""<<endl;
    // hydrostatic_pressure(items,Boussi_flag,neib,pressure,rho,f,posz) ;
    for(i=0;i<items.num_calc;i++){
        for(k=0;k<items.num_velocity;k++){
            float tmp = (vel_x[i]*items.cx[k] + vel_y[i]*items.cy[k] + vel_z[i]*items.cz[k])/pow(items.c,2) ;
            f[i*items.num_velocity+k] = items.weight[k]*(pressure[i]+3.0*tmp+4.5*pow(tmp,2)-1.5*(pow(vel_x[i],2)+pow(vel_y[i],2)+pow(vel_z[i],2))/pow(items.c,2)) ;
            g[i*items.num_velocity+k] = items.weight[k]*phi[i]*(1.0+3.0*tmp+4.5*pow(tmp,2)-1.5*(pow(vel_x[i],2)+pow(vel_y[i],2)+pow(vel_z[i],2))/pow(items.c,2)) ;
        }
    }

    // set IBM points
    items.num_IBMpoints = sqrt(pow(0.29,2)+pow(0.29/0.15,2))/items.dx ;
    vector<float> velB, posB, angleV_B, quaternion, quaS, IB, massB, FB, Torque, densB ;
    vector<int> num_IBMpoints, lattice_id ;
    vector<float> posw, Gw, velw, oposw, onB_vec, nB_vec ;
    // decide IB infomation
    num_IBMpoints.push_back(items.num_IBMpoints) ;
    posB.push_back(0) ; 
    posB.push_back(items.dx*items.ny/2.0) ; 
    posB.push_back(0) ;
    quaternion.push_back(1); 
    for(i=0;i<3;i++){
        quaternion.push_back(0); velB.push_back(0) ; angleV_B.push_back(0) ; Torque.push_back(0) ; FB.push_back(0);
    }
    for(i=0;i<9;i++){quaS.push_back(0);}
    set_quaternionS(0,quaternion[0],quaternion[1],quaternion[2],quaternion[3],quaS) ;
    densB.push_back(1.0*1000) ; massB.push_back(densB[0]) ; // density times area(2D)
    IB.push_back(massB[0]) ; IB.push_back(IB[0]) ; IB.push_back(IB[0]) ;

    cout<<"dens="<<densB[0]<<" massB="<<massB[0]<<endl;

    cout<<"set each IB points"<<endl;
    for(k=0;k<items.num_IBMpoints;k++){
        int near_id=0 ;
        oposw.push_back(20.0/sqrt(409.0)*items.dx*k) ;
        oposw.push_back(0.5*items.dx) ; // y
        oposw.push_back(items.nz*items.dx - 0.29/(items.num_IBMpoints-1.0)*k ) ; 
        // 楕円の法線ベクトルを算出　原点を0とする楕円の法線ベクトル成分は(2x/a^2 , 2y/b^2)
        onB_vec.push_back(3.0/sqrt(409.0)) ;
        onB_vec.push_back(0) ; // y
        onB_vec.push_back(20.0/sqrt(409.0)) ;
        for(i=0;i<3;i++){ 
            Gw.push_back(0) ; velw.push_back(0) ;
            nB_vec.push_back(onB_vec[k*3+i]) ; posw.push_back(oposw[k*3+i]) ;
        }
        float dist1 = 100 ;
        for(i=0;i<items.num_calc;i++){
            float dist2 = sqrt(pow(posx[i]-posw[k*3+0],2) +pow(posy[i]-posw[k*3+1],2) +pow(posz[i]-posw[k*3+2],2) ) ;
            if(dist1>dist2){
                dist1 = dist2 ; near_id = i ;
            }
        }
        lattice_id.push_back(near_id) ;
    }
    cout<<" number of IBM Points = "<<items.num_IBMpoints<< " nz= "<< items.nz<<endl ;

    // index show in spreadsheet
    // https://docs.google.com/spreadsheets/d/1wy2RkS1ECD7LtZCgyQAtm0fKckvEZmfZUeOMNmJwrgk/edit?gid=0#gid=0
    item.push_back(items.dx) ; item.push_back(items.dt) ; item.push_back(items.c) ; 
    item.push_back(items.nx) ; item.push_back(items.ny) ; item.push_back(items.nz) ; item.push_back(items.num_velocity) ; // 0~6
    item.push_back(items.ratiox) ; item.push_back(items.ratioy) ; item.push_back(items.PFthick) ; // 7~9
    // num_calc num_wall
    item.push_back(items.num_calc) ; item.push_back(rho.size()-items.num_calc) ;
    // num_IBM_points IBMdx
    item.push_back(items.num_IBMpoints) ; item.push_back(items.dx/2.0) ;
    item.push_back(items.nu) ; item.push_back(pow(10,-9)) ; item.push_back(items.sigma) ;
    item.push_back(items.tau) ; item.push_back(items.taus) ;
    // wall function用の変数を準備
    vector<int> wall1, wall2, wall3, wall4, wall5, wall6 ;
    set_walln(item, neib, wall1, wall2, wall3, wall4, wall5, wall6) ;
    item.push_back(wall1.size()) ; item.push_back(wall2.size()) ; item.push_back(wall3.size()) ; 
    item.push_back(wall4.size()) ; item.push_back(wall5.size()) ; item.push_back(wall6.size()) ;

    for(i=0;i<items.num_velocity;i++){item.push_back(items.weight[i]) ;}
    for(i=0;i<items.num_velocity;i++){item.push_back(items.cx[i]) ;}
    for(i=0;i<items.num_velocity;i++){item.push_back(items.cy[i]) ;}
    for(i=0;i<items.num_velocity;i++){item.push_back(items.cz[i]) ;}

    if     (items.num_velocity==9 ){set_bound2D(item, items.num_calc, neib, nextK, nextB) ;printf("call set_bound2D\n") ;}
    else if(items.num_velocity==27){set_bound3D(item, items.num_calc, neib, nextK, nextB) ;printf("call set_bound3D\n") ;}

    printf("allocate device memory \n");
    int *d_neib, *d_nextB, *d_nextK ;
    float *d_f, *d_ftmp, *d_fout, *d_feq, *d_g ;
    float *d_posx, *d_posy, *d_posz, *d_delX, *d_delY ;
    float *d_rho, *d_u, *d_v, *d_w, *d_sal, *d_phi, *d_pressure, *d_tau, *d_taus ;
    float *d_phiold, *d_uold, *d_vold, *d_wold ;
    float *d_Fk, *d_Fx, *d_Fy, *d_Fz, *d_FIBx, *d_FIBy, *d_FIBz ;
    float *d_items, *d_M, *d_Minv, *d_S, *d_MM; 
    int *d_wall1, *d_wall2, *d_wall3, *d_wall4, *d_wall5, *d_wall6 ;
    // IBM
    int   *d_lattice_id ; 
    float *d_velB, *d_posB ;
    float *d_angleVB, *d_quaternion, *d_quaS, *d_IB, *d_massB ;
    float *d_FB, *d_Torque, *d_densB, *d_posw, *d_oposw, *d_Gw, *d_velw, *d_nBvec, *d_onBvec ;
    float *d_quatold ;
    cuMallocCopy(&d_neib, neib) ; cuMallocCopy(&d_nextB, nextB) ; cuMallocCopy(&d_nextK,nextK) ;
    cuMallocCopy(&d_M, M)        ; cuMallocCopy(&d_MM, MM) ; 
    cuMallocCopy(&d_Minv, M_inv) ; cuMallocCopy(&d_S, S) ; 
    cuMallocCopy(&d_items,item) ;
    cuMallocCopy(&d_f, f) ;  cuMallocCopy(&d_ftmp, f) ; cuMallocCopy(&d_fout, f) ; cuMallocCopy(&d_feq, Fk) ; cuMallocCopy(&d_Fk, Fk) ;
    cuMallocCopy(&d_g, g) ;  
    cuMallocCopy(&d_pressure,pressure) ; cuMallocCopy(&d_sal,sal) ; cuMallocCopy(&d_phi,phi) ; 
    cuMallocCopy(&d_tau,tau) ; cuMallocCopy(&d_taus,taus) ;    
    cuMallocCopy(&d_Fx,Fx) ; cuMallocCopy(&d_Fy,Fy)   ; cuMallocCopy(&d_Fz,Fz) ;
    cuMallocCopy(&d_FIBx,Fx) ; cuMallocCopy(&d_FIBy,Fy)   ; cuMallocCopy(&d_FIBz,Fz) ;
    cuMallocCopy(&d_rho,rho) ; cuMallocCopy(&d_posx,posx) ; cuMallocCopy(&d_posy,posy) ; cuMallocCopy(&d_posz,posz) ;
    cuMallocCopy(&d_delX, delX) ; cuMallocCopy(&d_delY,delY) ;
    cuMallocCopy(&d_u,vel_x) ; cuMallocCopy(&d_v,vel_y)   ; cuMallocCopy(&d_w,vel_z) ;
    cuMallocCopy(&d_phiold,phi) ; cuMallocCopy(&d_uold,vel_x) ; cuMallocCopy(&d_vold,vel_y) ; cuMallocCopy(&d_wold,vel_z) ;
    cuMallocCopy(&d_wall1, wall1) ; cuMallocCopy(&d_wall2, wall2) ; cuMallocCopy(&d_wall3, wall3) ;
    cuMallocCopy(&d_wall4, wall4) ; cuMallocCopy(&d_wall5, wall5) ; cuMallocCopy(&d_wall6, wall6) ;
    // ibm
    cuMallocCopy(&d_lattice_id,lattice_id) ; cuMallocCopy(&d_velB,velB) ; cuMallocCopy(&d_posB,posB) ; 
    cuMallocCopy(&d_angleVB,angleV_B) ; cuMallocCopy(&d_quaternion,quaternion) ; cuMallocCopy(&d_quaS,quaS) ;
    cuMallocCopy(&d_IB,IB) ; cuMallocCopy(&d_massB,massB) ; cuMallocCopy(&d_FB,FB) ; 
    cuMallocCopy(&d_Torque,Torque) ; cuMallocCopy(&d_densB,densB) ; cuMallocCopy(&d_posw,posw) ; 
    cuMallocCopy(&d_oposw,oposw) ; cuMallocCopy(&d_Gw,Gw) ; cuMallocCopy(&d_velw,velw) ; 
    cuMallocCopy(&d_nBvec,nB_vec) ; cuMallocCopy(&d_onBvec,onB_vec) ; cuMallocCopy(&d_quatold,quaternion) ; 

    //////////////////////////////////////////////////////////////////////////////////////////////////
    output(item,posx,posy,posz,delX,delY,pressure,vel_x,vel_y,vel_z,sal,phi,rho,Fx,Fy,Fz,0,items.save_interval) ;
    IB_csv(0,item, posw, velw, Gw) ;
    printf("start main calculation \n");
    int blockSize = 64;
    int numBlocks = (rho.size() + blockSize - 1) / blockSize ;     
    auto start=chrono::high_resolution_clock::now() ;
    for(int timestep=1 ; timestep<items.total_count+1 ; timestep++){
        // velocity field
        // wall_function <float> <<<numBlocks, blockSize>>>(d_items, d_delX, d_delY, 1, 0, 0, wall1.size(), d_wall1, d_v, d_w, d_u, d_Fy, d_Fz, d_rho) ;
        // wall_function <float> <<<numBlocks, blockSize>>>(d_items, d_delX, d_delY, 1, 0, 0, wall3.size(), d_wall3, d_v, d_w, d_u, d_Fy, d_Fz, d_rho) ;
        // wall_function <float> <<<numBlocks, blockSize>>>(d_items, d_delX, d_delY, 0, 0, 1, wall5.size(), d_wall5, d_u, d_v, d_w, d_Fx, d_Fy, d_rho) ;
        // wall_function <float> <<<numBlocks, blockSize>>>(d_items, d_delX, d_delY, 0, 0, 1, wall6.size(), d_wall6, d_u, d_v, d_w, d_Fx, d_Fy, d_rho) ;
        // wall_function <float> <<<numBlocks, blockSize>>>(d_items, d_delX, d_delY, 0, 1, 0, wall2.size(), d_wall2, d_u, d_w, d_v, d_Fx, d_Fz, d_rho) ;
        // wall_function <float> <<<numBlocks, blockSize>>>(d_items, d_delX, d_delY, 0, 1, 0, wall4.size(), d_wall4, d_u, d_w, d_v, d_Fx, d_Fz, d_rho) ; // */
        equ_f         <float> <<<numBlocks, blockSize>>>(d_items, d_feq, d_pressure, d_u, d_v, d_w) ;
        Force         <float> <<<numBlocks, blockSize>>>(d_items, Boussi_flag, d_neib, d_f, d_feq, d_tau, d_Fk, d_Fx, d_Fy, d_Fz, d_pressure, d_rho, d_sal, d_phi, d_u, d_v, d_w, d_delX, d_delY, d_posx, d_posy, d_posz) ;
        col_f_MRT     <float> <<<numBlocks, blockSize>>>(d_items, d_tau, d_f, d_ftmp, d_feq, d_Fk, d_M, d_Minv, d_S, d_MM) ;
        IP_process(d_items,numBlocks,blockSize,d_neib,d_f,d_feq,d_ftmp,d_fout,d_nextB,d_nextK,d_posx,d_posy,d_delX,d_delY,0) ; // 0 => slip ; 1 => bounce back noslip

        // salinity 
        col_g_reg     <float> <<<numBlocks, blockSize>>>(d_items, d_taus, d_g, d_ftmp, d_feq, d_sal, d_u, d_v, d_w) ;
        IP_process(d_items,numBlocks,blockSize,d_neib,d_g,d_feq,d_ftmp,d_fout,d_nextB,d_nextK,d_posx,d_posy,d_delX,d_delY,0) ;
        // Phase Field
        /*col_PF        <float>       <<<numBlocks, blockSize>>>(d_items, d_neib, d_taus, d_g, d_gtmp, d_geq, d_phi, d_u, d_v, d_w, d_phiold, d_uold, d_vold, d_wold, d_posx, d_posy, d_posz) ;
        IP_process(d_items,numBlocks,blockSize,d_neib,d_g,d_feq,d_ftmp,d_fout,d_items_adv,d_nextB,d_nextK,d_posx,d_posy,d_delX,d_delY,0) ; // */
        
        update_scalar <float> <<<numBlocks, blockSize>>>(d_items, d_g, d_sal) ;
        // update_scalar <float> <<<numBlocks, blockSize>>>(d_items, d_, d_phi) ;
        update_rho    <float> <<<numBlocks, blockSize>>>(d_items, rhoL, rhoH, d_f, d_Fx, d_Fy, d_Fz, d_pressure, d_sal, d_phi, d_rho, d_u, d_v, d_w) ; 
        LES           <float> <<<numBlocks, blockSize>>>(d_items, d_neib, d_tau, d_taus, d_phi, d_rho, muL, muH, d_u, d_v, d_w, d_posx, d_posy, d_posz) ;

        // resetF<float><<<numBlocks, blockSize>>>(d_items, d_Fx, d_Fy, d_Fz, Fx.size()) ;
        for(i=0;i<1;i++){
            // SPM_ellipse3D         <<<numBlocks, blockSize>>>(d_items,d_f,d_posx,d_posy,d_posz,d_u,d_v,d_w,d_velB) ;
            // get_IBMGw2    <float> <<<numBlocks, blockSize>>>(d_items,d_lattice_id,d_neib,d_f,d_tau,d_posx,d_posy,d_posz,d_posw,d_posB,d_nBvec,d_u,d_v,d_w,d_velw,d_Fx,d_Fy,d_Fz,d_Gw,rhoH) ;
            // update_velIBM <float> <<<numBlocks, blockSize>>>(d_items,d_lattice_id,d_f,d_ftmp,d_pressure,d_tau,d_u,d_v,d_w,d_uold,d_vold,d_wold,d_Fx,d_Fy,d_Fz) ;

            get_IBMGw2    <float> <<<numBlocks, blockSize>>>(d_items,d_lattice_id,d_neib,d_f,d_tau,d_posx,d_posy,d_posz,d_posw,d_posB,d_nBvec,d_u,d_v,d_w,d_velw,d_FIBx,d_FIBy,d_FIBz,d_Gw,rhoH) ;
            update_velIBM2<float> <<<numBlocks, blockSize>>>(d_items,d_f,d_u,d_v,d_w,d_FIBx,d_FIBy,d_FIBz) ;
        } //
        set_wall_rho  <float> <<<numBlocks, blockSize>>>(d_items, d_neib, d_rho) ;
        set_wall_rho  <float> <<<numBlocks, blockSize>>>(d_items, d_neib, d_phi) ;
        // set_wall_rho  <float> <<<numBlocks, blockSize>>>(d_items, d_neib, d_u) ;  set_wall_rho  <float> <<<numBlocks, blockSize>>>(d_items, d_neib, d_v) ; set_wall_rho<float><<<numBlocks, blockSize>>>(d_items, d_neib, d_w) ;
        // CUDAのエラーをcheck
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
            return 1;
        } // */
        if(timestep%items.save_interval==0){
            hipMemcpy(pressure.data(), d_pressure , pressure.size()* sizeof(float), hipMemcpyDeviceToHost) ;
            hipMemcpy(vel_x.data(), d_u    ,  vel_x.size() * sizeof(float),         hipMemcpyDeviceToHost) ;
            hipMemcpy(vel_y.data(), d_v    ,  vel_y.size() * sizeof(float),         hipMemcpyDeviceToHost) ;
            hipMemcpy(vel_z.data(), d_w    ,  vel_z.size() * sizeof(float),         hipMemcpyDeviceToHost) ;
            hipMemcpy(Fx.data()   , d_Fx   ,  Fx.size()    * sizeof(float),         hipMemcpyDeviceToHost) ;
            hipMemcpy(Fy.data()   , d_Fy   ,  Fy.size()    * sizeof(float),         hipMemcpyDeviceToHost) ;
            hipMemcpy(Fz.data()   , d_Fz   ,  Fz.size()    * sizeof(float),         hipMemcpyDeviceToHost) ;
            hipMemcpy(sal.data()  , d_sal  ,  sal.size()   * sizeof(float),         hipMemcpyDeviceToHost) ;
            hipMemcpy(phi.data()  , d_phi  ,  phi.size()   * sizeof(float),         hipMemcpyDeviceToHost) ;
            hipMemcpy(rho.data()  , d_rho  ,  rho.size()   * sizeof(float),         hipMemcpyDeviceToHost) ;
            hipMemcpy(tau.data()  , d_tau  ,  tau.size()   * sizeof(float),         hipMemcpyDeviceToHost) ;
            printf("loop%02d  time=%f\n",timestep/items.save_interval,timestep*items.dt) ;
            output<float>(item,posx,posy,posz,delX,delY,pressure,vel_x,vel_y,vel_z,sal,phi,rho,Fx,Fy,Fz,timestep,items.save_interval) ;
            if(isnan(vel_x[0])!=0){
                cout<<"######################################"<<endl<<"Not a number is detected !"
                <<endl<<"######################################"<<endl; break;} // check NAN
            hipMemcpy(Gw.data()   , d_Gw   ,  Gw.size()    * sizeof(float),         hipMemcpyDeviceToHost) ;
            hipMemcpy(posw.data() , d_posw ,  posw.size()  * sizeof(float),         hipMemcpyDeviceToHost) ;
            hipMemcpy(velw.data() , d_velw ,  velw.size()  * sizeof(float),         hipMemcpyDeviceToHost) ;
            hipMemcpy(lattice_id.data(), d_lattice_id , lattice_id.size()* sizeof(int), hipMemcpyDeviceToHost) ;
            IB_csv(timestep/items.save_interval,item, posw, velw, Gw) ;
        }
        resetF<float><<<numBlocks, blockSize>>>(d_items, d_Fx, d_Fy, d_Fz, Fx.size()) ;
        resetF<float><<<numBlocks, blockSize>>>(d_items, d_FIBx, d_FIBy, d_FIBz, Fx.size()) ;
    }
    cout<<" dz= "<<items.dx<< " dt= " << items.dt<< " nu= "<<items.nu<<" tau= "<<items.tau<<endl;
    cout<<"taus= "<<items.taus<<" ratiox= "<<item[7]<<endl;
    cout<<"nx= "<<items.nx<< " ny= "<<items.ny<< " nz= "<<items.nz<<" num_velocity= "<<items.num_velocity<<endl;
    cout<<"number of calculation lattice is "<<items.num_calc<<" wall lattice is "<<rho.size()-items.num_calc<<endl; cout<<""<<endl;
    auto end=chrono::high_resolution_clock::now() ;
    chrono::duration<float> duration=end-start ;
    cout<<endl;
    cout<<" dz="<<items.dx<<"m, rx="<<items.ratiox<<", ry="<<items.ratioy<<
    ", nx="<<items.nx<<", ny="<<items.ny<<", nz="<<items.nz<<", dt="<<items.dt<<endl;
    /*for(i=50*items.nz;i<50*items.nz+3;i++){
        cout<<i<<" pressure="<<pressure[i]<<endl;
        for(k=0;k<items.num_velocity;k++){
            cout<<k<<" f="<<f[i*items.num_velocity+k]<< "  eq="<<pressure[i]*items.weight[k]<<" Delta="
            <<f[i*items.num_velocity+k]-pressure[i]*items.weight[k]<<" Fk="<<Fk[i*items.num_velocity+k]<< endl;
        }
        cout<<endl;
    } // */
    cout<<"###############################################"<<endl;
    cout<<"compute time = " << duration.count() <<endl;
    cout<<"###############################################"<<endl;

}